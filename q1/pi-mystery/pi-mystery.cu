// source: http://cacs.usc.edu/education/cs596/src/cuda/pi.cu

// Using CUDA device to calculate pi

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>

#define CSV_OUTPUT

int NBIN = 4096;

#define NUM_BLOCK  256  // Number of thread blocks
#define NUM_THREAD  256  // Number of threads per block
#define PI 3.1415926535  // known value of pi

#ifdef DP
	typedef double real_t;
#else
	typedef float real_t;
#endif



// Kernel that executes on the CUDA device
__global__ void cal_pi(real_t *sum, int nbin, real_t step, int nthreads, int nblocks) {
	long int i;
	long int total_tries;
	real_t x;

	total_tries = nbin*nthreads*nblocks;
	long int idx = blockIdx.x*blockDim.x+threadIdx.x;  // Sequential thread index across the blocks
	for (i=idx; i< total_tries; i+=nthreads*nblocks) {
		x = (i+0.5)*step;
		sum[idx] += 4.0/(1.0+x*x);
	}
}

real_t host_monte_carlo(long trials) {
	real_t x, y;
	long points_in_circle;
	for(long i = 0; i < trials; i++) {
		x = rand() / (float) RAND_MAX;
		y = rand() / (float) RAND_MAX;
		points_in_circle += (x*x + y*y <= 1.0f);
	}
	return 4.0f * points_in_circle / trials;
}

// Main routine that executes on the host
int main(int argc,char *argv[]) {
	clock_t start, stop;
	
	int tid;

	if(argc >1){
		NBIN = atoi(argv[1]);
	}

	dim3 dimGrid(NUM_BLOCK,1,1);  // Grid dimensions
	dim3 dimBlock(NUM_THREAD,1,1);  // Block dimensions
	real_t *sumHost, *sumDev;  // Pointer to host & device arrays

	#ifdef CSV_OUTPUT
			printf("[MYSTERY],precision,nbins,blocks,threads/block,gpu-pi-time,cpu-pi-time,gpu-pi,gpu-error,cpu-pi,cpu-error,\n");
			printf("[MYSTERY],");

		#ifdef DP
			printf("dp,");
		#else
			printf("sp,");
		#endif
			printf("%d,",NBIN);
			printf("%d,",NUM_BLOCK);
			printf("%d,",NUM_THREAD);
	#else
		printf("# of trials per thread = %d, # of blocks = %d, # of threads/block = %d.\n", NBIN,NUM_BLOCK, NUM_THREAD);
	#endif

	start = clock();

	real_t step = 1.0/NBIN;  // Step size
	size_t size = NUM_BLOCK*NUM_THREAD*sizeof(real_t);  //Array memory size
	sumHost = (real_t *)malloc(size);  //  Allocate array on host
	hipMalloc((void **) &sumDev, size);  // Allocate array on device
	// Initialize array in device to 0
	hipMemset(sumDev, 0, size);
	// Do calculation on device
	cal_pi <<<dimGrid, dimBlock>>> (sumDev, NBIN, step, NUM_THREAD, NUM_BLOCK); // call CUDA kernel
	// Retrieve result from device and store it in host array
	hipMemcpy(sumHost, sumDev, size, hipMemcpyDeviceToHost);

	real_t pi_gpu = 0.0f;

	for(tid=0; tid<NUM_THREAD*NUM_BLOCK; tid++){
		pi_gpu += sumHost[tid];
	}
	pi_gpu *= step/2;

	stop = clock();
	// Print results
	#ifdef CSV_OUTPUT
		printf("%f,",(stop-start)/(float)CLOCKS_PER_SEC);
	#else
		printf("GPU pi calculated in %f s.\n", (stop-start)/(float)CLOCKS_PER_SEC);
	#endif
	// Cleanup
	free(sumHost); 
	hipFree(sumDev);

	start = clock();
	real_t pi_cpu = host_monte_carlo(NUM_BLOCK * NUM_THREAD * NBIN);
	stop = clock();
	
	#ifdef CSV_OUTPUT
		printf("%f,",(stop-start)/(float)CLOCKS_PER_SEC);
	#else
		printf("CPU pi calculated in %f s.\n", (stop-start)/(float)CLOCKS_PER_SEC);
	#endif

	#ifdef CSV_OUTPUT
			printf("%f,",pi_gpu);
			printf("%f,",pi_gpu - PI);
			printf("%f,",pi_cpu);
			printf("%f,\n",pi_cpu - PI);
			
	#else

		printf("CUDA estimate of PI = %f [error of %f]\n", pi_gpu, pi_gpu - PI);
		printf("CPU estimate of PI = %f [error of %f]\n", pi_cpu, pi_cpu - PI);
	#endif

	return 0;
}
