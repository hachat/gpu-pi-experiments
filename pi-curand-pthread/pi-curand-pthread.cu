#include "hip/hip_runtime.h"
// Source: http://web.mit.edu/pocky/www/cudaworkshop/MonteCarlo/Pi.cu

// Written by Barry Wilkinson, UNC-Charlotte. Pi.cu  December 22, 2010.
//Derived somewhat from code developed by Patrick Rogers, UNC-C

#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <time.h>
#include <hiprand/hiprand_kernel.h>
#include <pthread.h>

#define TRIALS_PER_THREAD 4096
#define BLOCKS 256
#define THREADS 256
#define PI 3.1415926535  // known value of pi

__global__ void gpu_monte_carlo(float *estimate, hiprandState *states) {
	unsigned int tid = threadIdx.x + blockDim.x * blockIdx.x;
	int points_in_circle = 0;
	float x, y;

	hiprand_init(1234, tid, 0, &states[tid]);  // 	Initialize CURAND


	for(int i = 0; i < TRIALS_PER_THREAD; i++) {
		x = hiprand_uniform (&states[tid]);
		y = hiprand_uniform (&states[tid]);
		points_in_circle += (x*x + y*y <= 1.0f); // count if x & y is in the circle.
	}
	estimate[tid] = 4.0f * points_in_circle / (float) TRIALS_PER_THREAD; // return estimate of pi
}

float host_monte_carlo(long trials) {
	float x, y;
	long points_in_circle;
	for(long i = 0; i < trials; i++) {
		x = rand() / (float) RAND_MAX;
		y = rand() / (float) RAND_MAX;
		points_in_circle += (x*x + y*y <= 1.0f);
	}
	return 4.0f * points_in_circle / trials;
}

typedef enum{
	PRNG,
	RAND,
}random_generator_t;

typedef struct{
	int thread_id;
	long int ncount;
	random_generator_t rng_type;
	float estimate;
}try_arg_t;



void * parallel_monte_carlo_try(void * arg){
	struct prng *g;
	try_arg *args = (try_arg *)arg;
	long trials = args->ncount;
	int t = args->thread_id;
	
	if(args->rng_type == PRNG){
		g = prng_new("eicg(2147483647,111,1,0)");
		if(g == NULL){
			printf("Initializing random number generator failed\n");
			pthread_exit(NULL);
		}
	}
	//printf("Thread %d doing %d tries\n",t,ncount);

	float x, y;
	long points_in_circle = 0;
	for(long i = 0; i < trials; i++) {
		if(args->rng_type == RAND){
			x = rand() / (float) RAND_MAX;
			y = rand() / (float) RAND_MAX;
		}
		else if(args->rng_type == PRNG){
			x = prng_get_next(g);
			y = prng_get_next(g);
		}
		points_in_circle += (x*x + y*y <= 1.0f);
	}
	args->estimate = 4.0f * points_in_circle / trials;

	if(args->rng_type == PRNG){
		prng_reset(g);
		prng_free(g);
	}
	pthread_exit(NULL);	 
}


float host_pthread_monte_carlo(long trials,int num_pthreads,random_generator_t rng_type){
	long tries_per_pthread = 0;
	long t;
	int rc;
	pthread_t *threads;
	pthread_attr_t attr;
	try_arg_t *try_args;
	void * status;
	float pi_pthreads;

	try_args = (float *)malloc(num_pthreads*sizeof(try_arg_t));
	threads = (float *)malloc(num_pthreads*sizeof(pthread_t));  //  Allocate pthreads
	pthread_attr_init(&attr);
	pthread_attr_setdetachstate(&attr,PTHREAD_CREATE_JOINABLE);
	tries_per_pthread = BLOCKS * THREADS * TRIALS_PER_THREAD / num_pthreads;

	for(t = 0; t < num_pthreads; t++){
		try_args[t].thread_id = t;
		try_args[t].ncount = tries_per_pthread;
		try_args[t].rng_type = rng_type;
		try_args[t].estimate = 0;//For output
		rc = pthread_create(&threads[t],&attr,try,(void *)&tries_per_pthread);
		if(rc){
			printf("ERROR; return code from pthread_create()\
				 is %d\n", rc);
			exit(-1);
		}
	}
	pthread_attr_destroy(&attr);

	pi_pthreads = 0.0f;
	for(t = 0; t < num_pthreads; t++){
		rc = pthread_join(threads[t], &status);
		pi_pthreads += try_args[t].estimate;
	}

	free(try_args);
	free(threads);
	return pi_pthreads;
}

int main (int argc, char *argv[]) {
	clock_t start, stop;
	float host[BLOCKS * THREADS];
	float *dev;
	hiprandState *devStates;

	
	printf("# of trials per thread = %d, # of blocks = %d, # of threads/block = %d.\n", TRIALS_PER_THREAD,
BLOCKS, THREADS);

	start = clock();

	hipMalloc((void **) &dev, BLOCKS * THREADS * sizeof(float)); // allocate device mem. for counts
	
	hipMalloc( (void **)&devStates, THREADS * BLOCKS * sizeof(hiprandState) );

	gpu_monte_carlo<<<BLOCKS, THREADS>>>(dev, devStates);

	hipMemcpy(host, dev, BLOCKS * THREADS * sizeof(float), hipMemcpyDeviceToHost); // return results 

	float pi_gpu;
	for(int i = 0; i < BLOCKS * THREADS; i++) {
		pi_gpu += host[i];
	}

	pi_gpu /= (BLOCKS * THREADS);

	stop = clock();

	printf("GPU pi calculated in %f s.\n", (stop-start)/(float)CLOCKS_PER_SEC);

	start = clock();
	float pi_cpu = host_monte_carlo(BLOCKS * THREADS * TRIALS_PER_THREAD);
	stop = clock();
	printf("CPU pi calculated in %f s.\n", (stop-start)/(float)CLOCKS_PER_SEC);

	float pi_cpu_pthread;
	int num_pthreads = 0;
	random_generator_t rng_type = RAND;
	if(argc >1){
		num_pthreads = atoi(argv[1]);
		if(argc >2){
			if(strcmp(argv[2],"RAND")){
				rng_type = RAND;
			}else if(strcmp(argv[2],"PRNG")){
				rng_type = PRNG;
			}else{
				rng_type = RAND;
			}
		}
		start = clock();
		pi_cpu_pthread = host_pthread_monte_carlo(BLOCKS * THREADS * TRIALS_PER_THREAD,num_pthreads,rng_type);
		stop = clock();
		printf("CPU Pthread pi calculated in %f s. Used %d threads.\n", (stop-start)/(float)CLOCKS_PER_SEC,num_pthreads);
	}


	printf("CUDA estimate of PI = %f [error of %f]\n", pi_gpu, pi_gpu - PI);
	printf("CPU estimate of PI = %f [error of %f]\n", pi_cpu, pi_cpu - PI);
	printf("CPU pthread estimate of PI = %f [error of %f]\n", pi_cpu_pthread, pi_cpu_pthread - PI);
	
	return 0;
}

